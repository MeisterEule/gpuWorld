#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "memoryManager.hpp"
#include "grid_utils.h"
#include "random.h"

__global__ void setup_curand_kernel (hiprandState *state, uint64_t seed) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &state[tid]);
}

__global__ void fill_array_kernel (int *data, int N, int min, int max, int stride, hiprandState *globalState) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= N) return;
	hiprandState localState = globalState[tid];

	tid *= stride;
	for (int i = 0; i < stride && tid + i < N; i++) {
	   double f = (double)hiprand_uniform(&localState);
 	   data[tid + i] = min + (int)(f * (max - min));
	}
}

cudaRNG::cudaRNG (size_t bytes, uint64_t init_seed) {
	reserved_bytes = bytes;
	seed = init_seed;
	gen_stride = 0;
	n_threads = 0;
	n_blocks = 0;
}

void cudaRNG::initRNG (memoryManager *mm, int N_numbers) {
	int n_curand_states = reserved_bytes / sizeof(hiprandState) + 1;
	getGridDimension1D (n_curand_states, &n_blocks, &n_threads);
	gen_stride = N_numbers / n_curand_states;
	if (gen_stride == 0) gen_stride = 1;
        mm->deviceAllocate<hiprandState>(deviceCurandStates, n_threads * n_blocks, "curandStates");
        setup_curand_kernel<<<n_blocks,n_threads>>>(deviceCurandStates, seed);
}

int *cudaRNG::generate (memoryManager *mm, int N_numbers, int min, int max) {
   int *rng_data_h = (int*)malloc(N_numbers * sizeof(int));
   int *rng_data_d;
   mm->deviceAllocate<int> (rng_data_d, N_numbers, "devRandomNumbers");

   fill_array_kernel<<<n_blocks,n_threads>>>(rng_data_d, N_numbers, min, max, gen_stride, deviceCurandStates);
   hipMemcpy(rng_data_h, rng_data_d, N_numbers * sizeof(int), hipMemcpyDeviceToHost);
   mm->deviceFree (rng_data_d);
   return rng_data_h;
}

void cudaRNG::freeRNG (memoryManager *mm) {
   mm->deviceFree<hiprandState>(deviceCurandStates);
}

void cudaRNG::printStatus () {
	printf ("cudaRNG: \n");
	printf ("  reserved bytes: %d\n", reserved_bytes);
	printf ("  seed: %lld\n", seed);
	printf ("  n_blocks: %d\n", n_blocks);
	printf ("  n_threads: %d\n", n_threads);
	printf ("  stride: %d\n", gen_stride);
}
