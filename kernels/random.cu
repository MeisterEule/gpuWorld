#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "grid_utils.h"

__global__ void setup_curand_kernel (hiprandState *state, uint64_t seed) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &state[tid]);
}

__global__ void fill_array_kernel (int *data, int N, int min, int max, hiprandState *globalState) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= N) return;
	hiprandState localState = globalState[tid];

 	data[tid] = min + (int)(hiprand_uniform(&localState) * (max - min));
}

static hiprandState *deviceCurandStates;
static int curand_threads;
static int curand_blocks;


void initRNG (uint64_t seed, int N_numbers) {
   getGridDimension1D (N_numbers, &curand_blocks, &curand_threads);
   hipMalloc(&deviceCurandStates, curand_threads * curand_blocks * sizeof(hiprandState));
   setup_curand_kernel<<<curand_blocks,curand_threads>>>(deviceCurandStates, seed);
}

int *generateRandomArrayInt (int N, int min, int max) {
   int *rng_data_h = (int*)malloc(N * sizeof(int));
   int *rng_data_d;
   hipMalloc((void**)&rng_data_d, N * sizeof(int));
   fill_array_kernel<<<curand_blocks,curand_threads>>>(rng_data_d, N, min, max, deviceCurandStates);
   hipMemcpy(rng_data_h, rng_data_d, N * sizeof(int), hipMemcpyDeviceToHost);
   hipFree(rng_data_d);
   return rng_data_h;
}
