#include "hip/hip_runtime.h"
#include "types.h"
#include "memoryManager.hpp"
#include "grid_utils.h"
#include "spmv.hpp"

#include "random.h"

__global__ void spmv_coo_kernel (int *rowidx, int *colidx, float *values, float *v, float *w, LDIM nnz) {
 	LDIM idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nnz) return;

	int row = rowidx[idx];
	int col = colidx[idx];
	float value = values[idx];
	atomicAdd(&w[row], v[col]*value);
}

__global__ void spmv_coo_kernel (LDIM *rowidx, LDIM *colidx, float *values, float *v, float *w, LDIM nnz) {
 	LDIM idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nnz) return;

	LDIM row = rowidx[idx];
	LDIM col = colidx[idx];
	float value = values[idx];
	atomicAdd(&w[row], v[col]*value);
}

void launch_spmv_coo_kernel (int n_blocks, int n_threads, int *rowidx, int *colidx, float *values, float *v, float *w, LDIM nnz) {
	spmv_coo_kernel<<<n_blocks,n_threads>>>(rowidx, colidx, values, v, w, nnz);
}

void launch_spmv_coo_kernel (int n_blocks, int n_threads, LDIM *rowidx, LDIM *colidx, float *values, float *v, float *w, LDIM nnz) {
	spmv_coo_kernel<<<n_blocks,n_threads>>>(rowidx, colidx, values, v, w, nnz);
}

