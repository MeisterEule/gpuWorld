#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "compute_step.hpp"
#include "memoryManager.hpp"
#include "grid_utils.h"

#define SAFE_CUDA(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    } \
} while (0)


__global__ void count_elements_in_array_kernel_int (int *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	atomicAdd(&(count[data[tid]]), 1);
}

__global__ void count_elements_in_array_kernel (char *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	int alphabet_position = data[tid] - 'a';
	if (alphabet_position >= 0 && alphabet_position < 26) {
	   atomicAdd(&(count[alphabet_position]), 1);
	}
}

__global__ void avg_atomic_kernel (int *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	atomicAdd(&(count[0]), data[tid]);
	__syncthreads();
}


void countElementsInArray (memoryManager *mm, ComputeStep<int> *cs_h) {
        int n_data_in = cs_h->n_data_in->front();
	int n_data_out = cs_h->n_data_out->front();
        bool input_on_device = cs_h->input_on_device->front();
	bool output_on_device = cs_h->input_on_device->front();
	int *data_in = cs_h->data_in->front();
        int *data_out = cs_h->data_out->front();
	int n_threads, n_blocks;
	getGridDimension1D (n_data_in, &n_blocks, &n_threads);


	int *data_d;
	if (input_on_device) {
		data_d = data_in;
	} else {
		mm->deviceAllocate<int>(data_d, n_data_in);
	 	hipMemcpy(data_d, data_in, n_data_in * sizeof(int), hipMemcpyHostToDevice);
	}

	int *count_d;
	if (output_on_device) {
		count_d = data_out;
	} else {
		mm->deviceAllocate<int>(count_d, n_data_out);
	}
	hipMemset(count_d, 0, n_data_out * sizeof(int));

	count_elements_in_array_kernel_int<<<n_blocks,n_threads>>>(data_d, count_d, n_data_in);

	if (!output_on_device) {
	   hipMemcpy(data_out, count_d, n_data_out * sizeof(int), hipMemcpyDeviceToHost);
	}
	if (!input_on_device) hipFree(data_d);
	if (!output_on_device) hipFree(count_d);
}

void computeAverageOfArray (ComputeStep<int> *cs_h) {
	// TODO: Check if N_out = 1
        int n_data_in = cs_h->n_data_in->front();
	int n_data_out = cs_h->n_data_out->front();
        bool input_on_device = cs_h->input_on_device->front();
	bool output_on_device = cs_h->input_on_device->front();
	int *data_in = cs_h->data_in->front();
        int *data_out = cs_h->data_out->front();

	int n_threads, n_blocks;
	getGridDimension1D (n_data_in, &n_blocks, &n_threads);

	int *data_d;
	if (input_on_device) {
		data_d = data_in;
	} else {
		hipMalloc((void**)&data_d, n_data_in * sizeof(int));
		hipMemcpy(data_d, data_in, n_data_in * sizeof(int), hipMemcpyHostToDevice);
	}

	int *count_d;
	if (output_on_device) {
		count_d = data_out;
	} else {
		hipMalloc((void**)&count_d, n_data_out * sizeof(int));
	}
	hipMemset(count_d, 0, n_data_out * sizeof(int));

	avg_atomic_kernel<<<n_blocks,n_threads>>>(data_d, count_d, n_data_in);

	if (!cs_h->output_on_device) {
	   hipMemcpy(data_out, count_d, n_data_out * sizeof(int), hipMemcpyDeviceToHost);
	}
	if (!input_on_device) hipFree(data_d);
	if (!output_on_device) hipFree(count_d);
}

int *countElementsInArray (char *data, int n_data) {
	int n_threads, n_blocks;
	getGridDimension1D (n_data, &n_blocks, &n_threads);

	int *count_h = (int*)malloc(n_data * sizeof(int));
	memset (count_h, 0, 26 * sizeof(int));

	char *data_d;
        int *count_d;
	hipMalloc((void**)&data_d, n_data * sizeof(char));
	hipMalloc((void**)&count_d, 26 * sizeof(int));
	hipMemcpy(data_d, data, n_data * sizeof(char), hipMemcpyHostToDevice);
	hipMemset(count_d, 0, n_data * sizeof(int));

	count_elements_in_array_kernel<<<n_blocks,n_threads>>>(data_d, count_d, n_data);

	hipMemcpy(count_d, count_h, n_data * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(count_d);
	hipFree(data_d);

	return count_h;
}
