#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "grid_utils.h"

#define SAFE_CUDA(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    } \
} while (0)


__global__ void count_elements_in_array_kernel_int (int *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	atomicAdd(&(count[data[tid]]), 1);
}

__global__ void count_elements_in_array_kernel (char *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	int alphabet_position = data[tid] - 'a';
	if (alphabet_position >= 0 && alphabet_position < 26) {
	   atomicAdd(&(count[alphabet_position]), 1);
	}
}


int *countElementsInArray (int *data, int n_data) {
	printf ("HUHU\n");
	int n_threads, n_blocks;
	getGridDimension1D (n_data, &n_blocks, &n_threads);
        int mem_size = n_data * sizeof(int);

	int *count_h = (int*)malloc(mem_size);
	memset (count_h, 0, mem_size);

	int *data_d, *count_d;
	SAFE_CUDA(hipMalloc((void**)&data_d, mem_size));
	SAFE_CUDA(hipMalloc((void**)&count_d, mem_size));
	SAFE_CUDA(hipMemcpy(data_d, data, mem_size, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemset(count_d, 0, mem_size));

	count_elements_in_array_kernel_int<<<n_blocks,n_threads>>>(data_d, count_d, n_data);
	hipError_t ce = hipGetLastError();
	printf ("hipError_t: %s\n", hipGetErrorString(ce));

	SAFE_CUDA(hipMemcpy(count_h, count_d, mem_size, hipMemcpyDeviceToHost));
	SAFE_CUDA(hipFree(count_d));
	SAFE_CUDA(hipFree(data_d));

	return count_h;
}

int *countElementsInArray (char *data, int n_data) {
	int n_threads, n_blocks;
	getGridDimension1D (n_data, &n_blocks, &n_threads);

	int *count_h = (int*)malloc(n_data * sizeof(int));
	memset (count_h, 0, 26 * sizeof(int));

	char *data_d;
        int *count_d;
	hipMalloc((void**)&data_d, n_data * sizeof(char));
	hipMalloc((void**)&count_d, 26 * sizeof(int));
	hipMemcpy(data_d, data, n_data * sizeof(char), hipMemcpyHostToDevice);
	hipMemset(count_d, 0, n_data * sizeof(int));

	count_elements_in_array_kernel<<<n_blocks,n_threads>>>(data_d, count_d, n_data);

	hipMemcpy(count_d, count_h, n_data * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(count_d);
	hipFree(data_d);

	return count_h;
}
