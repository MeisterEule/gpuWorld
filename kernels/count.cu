#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "types.h"
#include "compute_step.hpp"
#include "memoryManager.hpp"
#include "grid_utils.h"

#define SAFE_CUDA(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    } \
} while (0)


__global__ void count_elements_in_array_kernel_int (int *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	atomicAdd(&(count[data[tid]]), 1);
}

__global__ void count_elements_in_array_kernel_char (char *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	if (data[tid] >= 0 && data[tid] < 128) {
	   atomicAdd(&(count[data[tid]]), 1);
	}
}

__global__ void avg_atomic_kernel (int *data, int *count, int n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	atomicAdd(&(count[0]), data[tid]);
	__syncthreads();
}

__global__ void count_nonzero_kernel (int *data, LDIM *count, LDIM n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	if (data[tid] > 0) atomicAdd(&(count[0]), 1);
}

__global__ void count_nonzero_kernel (float *data, LDIM *count, LDIM n_data) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n_data) return;
	if (data[tid] > 0) atomicAdd(&(count[0]), 1);
}


LDIM countNonzeros (memoryManager *mm, float *data, LDIM n_data, bool input_on_device) {
	int n_threads, n_blocks;
	getGridDimension1D (n_data, &n_blocks, &n_threads);

	float *data_d;
	if (input_on_device) {
		data_d = data;
	} else {
		mm->deviceAllocate<float>(data_d, n_data, "countInput");
	 	hipMemcpy(data_d, data, n_data * sizeof(int), hipMemcpyHostToDevice);
	}

	LDIM *count_d;
        mm->deviceAllocate<LDIM>(count_d, 1, "countOutput");
	hipMemset(count_d, 0, sizeof(LDIM));
	
	count_nonzero_kernel<<<n_blocks,n_threads>>>(data_d, count_d, n_data);

	LDIM count_h;
	hipMemcpy (&count_h, count_d, sizeof(LDIM), hipMemcpyDeviceToHost);

	if (!input_on_device) mm->deviceFree<float>(data_d);
	mm->deviceFree<LDIM>(count_d);

	return count_h;
}



long long countNonzeros (memoryManager *mm, ComputeStep<float,int> *cs) {
	int n_data = cs->n_data_in->front();
	bool input_on_device = cs->input_on_device->front();
	float *data = cs->data_in->front();
	return countNonzeros (mm, data, n_data, input_on_device);
}

void countElementsInArray (memoryManager *mm, ComputeStep<int,int> *cs_h) {
        int n_data_in = cs_h->n_data_in->front();
	int n_data_out = cs_h->n_data_out->front();
        bool input_on_device = cs_h->input_on_device->front();
	bool output_on_device = cs_h->input_on_device->front();
	int *data_in = cs_h->data_in->front();
        int *data_out = cs_h->data_out->front();
	int n_threads, n_blocks;
	getGridDimension1D (n_data_in, &n_blocks, &n_threads);


	int *data_d;
	if (input_on_device) {
		data_d = data_in;
	} else {
		mm->deviceAllocate<int>(data_d, n_data_in, "countInput");
	 	hipMemcpy(data_d, data_in, n_data_in * sizeof(int), hipMemcpyHostToDevice);
	}

	int *count_d;
	if (output_on_device) {
		count_d = data_out;
	} else {
		mm->deviceAllocate<int>(count_d, n_data_out, "countOutput");
	}
	hipMemset(count_d, 0, n_data_out * sizeof(int));

	count_elements_in_array_kernel_int<<<n_blocks,n_threads>>>(data_d, count_d, n_data_in);

	if (!output_on_device) {
	   hipMemcpy(data_out, count_d, n_data_out * sizeof(int), hipMemcpyDeviceToHost);
	}
	if (!input_on_device) mm->deviceFree<int>(data_d);
	if (!output_on_device) mm->deviceFree<int>(count_d);
}

int *countElementsInArray (memoryManager *mm, ComputeStep<char,int> *cs_h) {
        int n_data_in = cs_h->n_data_in->front();
	int n_data_out = cs_h->n_data_out->front();
	if (n_data_out != 128) {
		printf ("Character count output must have 128 elements (Nr. of symbols in ASCII).\n");
		return NULL;
	}
        bool input_on_device = cs_h->input_on_device->front();
	bool output_on_device = cs_h->input_on_device->front();
	char *data_in = cs_h->data_in->front();
        int *data_out = cs_h->data_out->front();
	int n_threads, n_blocks;
	getGridDimension1D (n_data_in, &n_blocks, &n_threads);

	char *data_d;
	if (input_on_device) {
		data_d = data_in;
	} else {
		mm->deviceAllocate<char>(data_d, n_data_in, "countInput");
	 	hipMemcpy(data_d, data_in, n_data_in * sizeof(char), hipMemcpyHostToDevice);
	}

	int *count_d;
	if (output_on_device) {
		count_d = data_out;
	} else {
		mm->deviceAllocate<int>(count_d, n_data_out, "countOutput");
	}
	hipMemset(count_d, 0, n_data_out * sizeof(int));

	count_elements_in_array_kernel_char<<<n_blocks,n_threads>>>(data_d, count_d, n_data_in);

	int *count_h = (int*)malloc(26 * sizeof(int));

	if (!output_on_device) hipMemcpy(count_h, count_d, n_data_out * sizeof(int), hipMemcpyDeviceToHost);
	if (!input_on_device) mm->deviceFree<int>(count_d);
	if (!output_on_device) mm->deviceFree<char>(data_d);

	return count_h;
}
